#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void printHello()
{
    printf("宋晓维提交\n");
    printf("Hello\n");
}

int main()
{
    printHello << <1, 8 >> > ();
    hipDeviceSynchronize(); 
    return 0;
}
